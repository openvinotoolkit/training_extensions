#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

#define THREADS_PER_BLOCK 1024

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 65000;
  return min(optimal_block_num, max_block_num);
}

template <typename scalar_t>
__global__ void ROIPoolForward(const int nthreads, const scalar_t *bottom_data,
                               const scalar_t *rois,
                               const scalar_t spatial_scale, const int channels,
                               const int height, const int width,
                               const int pooled_h, const int pooled_w,
                               scalar_t *top_data, int *argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_w;
    int ph = (index / pooled_w) % pooled_h;
    int c = (index / pooled_w / pooled_h) % channels;
    int n = index / pooled_w / pooled_h / channels;

    const scalar_t *offset_rois = rois + n * 5;
    int roi_batch_ind = offset_rois[0];
    // calculate the roi region on feature maps
    scalar_t roi_x1 = offset_rois[1] * spatial_scale;
    scalar_t roi_y1 = offset_rois[2] * spatial_scale;
    scalar_t roi_x2 = (offset_rois[3] + 1) * spatial_scale;
    scalar_t roi_y2 = (offset_rois[4] + 1) * spatial_scale;

    // force malformed rois to be 1x1
    scalar_t roi_w = roi_x2 - roi_x1;
    scalar_t roi_h = roi_y2 - roi_y1;
    if (roi_w <= 0 || roi_h <= 0) continue;

    scalar_t bin_size_w = roi_w / static_cast<scalar_t>(pooled_w);
    scalar_t bin_size_h = roi_h / static_cast<scalar_t>(pooled_h);

    // the corresponding bin region
    int bin_x1 = floor(static_cast<scalar_t>(pw) * bin_size_w + roi_x1);
    int bin_y1 = floor(static_cast<scalar_t>(ph) * bin_size_h + roi_y1);
    int bin_x2 = ceil(static_cast<scalar_t>(pw + 1) * bin_size_w + roi_x1);
    int bin_y2 = ceil(static_cast<scalar_t>(ph + 1) * bin_size_h + roi_y1);

    // add roi offsets and clip to input boundaries
    bin_x1 = min(max(bin_x1, 0), width);
    bin_y1 = min(max(bin_y1, 0), height);
    bin_x2 = min(max(bin_x2, 0), width);
    bin_y2 = min(max(bin_y2, 0), height);
    bool is_empty = (bin_y2 <= bin_y1) || (bin_x2 <= bin_x1);

    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int max_idx = -1;
    bottom_data += (roi_batch_ind * channels + c) * height * width;

    // Define an empty pooling region to be zero
    scalar_t max_val = is_empty ? static_cast<scalar_t>(0)
                                : bottom_data[bin_y1 * width + bin_x1] - 1;

    for (int h = bin_y1; h < bin_y2; ++h) {
      for (int w = bin_x1; w < bin_x2; ++w) {
        int offset = h * width + w;
        if (bottom_data[offset] > max_val) {
          max_val = bottom_data[offset];
          max_idx = offset;
        }
      }
    }
    top_data[index] = max_val;
    if (argmax_data != NULL) argmax_data[index] = max_idx;
  }
}

int ROIPoolForwardLaucher(const at::Tensor features, const at::Tensor rois,
                          const float spatial_scale, const int channels,
                          const int height, const int width, const int num_rois,
                          const int pooled_h, const int pooled_w,
                          at::Tensor output, at::Tensor argmax) {
  const int output_size = num_rois * channels * pooled_h * pooled_w;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      features.type(), "ROIPoolLaucherForward", ([&] {
        const scalar_t *bottom_data = features.data<scalar_t>();
        const scalar_t *rois_data = rois.data<scalar_t>();
        scalar_t *top_data = output.data<scalar_t>();
        int *argmax_data = argmax.data<int>();

        ROIPoolForward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                channels, height, width, pooled_h, pooled_w, top_data,
                argmax_data);
      }));
  THCudaCheck(hipGetLastError());
  return 1;
}

template <typename scalar_t>
__global__ void ROIPoolBackward(const int nthreads, const scalar_t *top_diff,
                                const scalar_t *rois, const int *argmax_data,
                                const scalar_t spatial_scale,
                                const int channels, const int height,
                                const int width, const int pooled_h,
                                const int pooled_w, scalar_t *bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_w;
    int ph = (index / pooled_w) % pooled_h;
    int c = (index / pooled_w / pooled_h) % channels;
    int n = index / pooled_w / pooled_h / channels;

    int roi_batch_ind = rois[n * 5];
    int bottom_index = argmax_data[(n * channels + c) * pooled_h * pooled_w +
                                   ph * pooled_w + pw];

    atomicAdd(bottom_diff + (roi_batch_ind * channels + c) * height * width +
                  bottom_index,
              top_diff[index]);
  }
}

int ROIPoolBackwardLaucher(const at::Tensor top_grad, const at::Tensor rois,
                           const at::Tensor argmax, const float spatial_scale,
                           const int batch_size, const int channels,
                           const int height, const int width,
                           const int num_rois, const int pooled_h,
                           const int pooled_w, at::Tensor bottom_grad) {
  const int output_size = num_rois * pooled_h * pooled_w * channels;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "ROIPoolLaucherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *rois_data = rois.data<scalar_t>();
        const int *argmax_data = argmax.data<int>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();

        if (sizeof(scalar_t) == sizeof(double)) {
          fprintf(stderr, "double is not supported\n");
          exit(-1);
        }

        ROIPoolBackward<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, argmax_data,
                scalar_t(spatial_scale), channels, height, width, pooled_h,
                pooled_w, bottom_diff);
      }));
  THCudaCheck(hipGetLastError());
  return 1;
}
